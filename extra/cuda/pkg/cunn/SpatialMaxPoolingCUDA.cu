
#include "SpatialMaxPoolingCUDA/spatial_pool_fprop.cu"
#include "SpatialMaxPoolingCUDA/spatial_pool_bprop.cu"

static int cunn_SpatialMaxPoolingCUDA_updateOutput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  // THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  luaL_argcheck(L, input->nDimension == 4, 2, "4D (batch) tensor expected");

  long nInputCols = input->size[2];
  long nInputRows = input->size[1];
  long nInputPlane = input->size[0];
  long batchSize = input->size[3];
  long nOutputCols = (nInputCols - kW) / dW + 1;
  long nOutputRows = (nInputRows - kH) / dH + 1;

  luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

  luaL_argcheck(L, THCudaTensor_isContiguous(input), 2, "input must be contiguous");
  float *input_data = THCudaTensor_data(input);
  
  THCudaTensor_resize4d(output, nInputPlane, nOutputRows, nOutputCols, batchSize);
  // THCudaTensor_resize5d(indices, 2, nInputPlane, nOutputRows, nOutputCols, batchSize);

  // float *indices_data = THCudaTensor_data(indices); // IGNORED !
  float *output_data = THCudaTensor_data(output);

  // kernel 
  spatialMaxPool_updateOutput<MaxPooler>
    (input_data, output_data, 
     nInputPlane, nInputRows, nInputCols, batchSize,
     nOutputRows, nOutputCols, 
     kH, kW,
     0, dW);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxPoolingCUDA.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int cunn_SpatialMaxPoolingCUDA_updateGradInput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  luaL_argcheck(L, dW == kW, 1, "dW and kW must be equal");

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  // THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  // TODO more checks ?
  // THCudaTensor_resize4d(output, nInputPlane, nOutputRows, nOutputCols, batchSize);

  long nInputCols = input->size[2];
  long nInputRows = input->size[1];
  long nInputPlane = input->size[0];
  long batchSize = input->size[3];
  long nOutputCols = (nInputCols - kW) / dW + 1;
  long nOutputRows = (nInputRows - kH) / dH + 1;

  THCudaTensor_resizeAs(gradInput, input);
  THCudaTensor_zero(gradInput);

  // float *indices_data = THCudaTensor_data(indices);
  float *input_data = THCudaTensor_data(input);
  float *output_data = THCudaTensor_data(output);
  float *gradOutput_data = THCudaTensor_data(gradOutput);
  float *gradInput_data = THCudaTensor_data(gradInput);

  // kernel 
  spatialMaxPool_updateGradInput
    (input_data, gradOutput_data, output_data, gradInput_data,
     nInputPlane, nInputRows, nInputCols, batchSize,
     nOutputRows, nOutputCols, 
     kH, kW,
     0, dW);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxPoolingCUDA.updateGradInput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static const struct luaL_Reg cunn_SpatialMaxPoolingCUDA__ [] = {
  {"SpatialMaxPoolingCUDA_updateOutput", cunn_SpatialMaxPoolingCUDA_updateOutput},
  {"SpatialMaxPoolingCUDA_updateGradInput", cunn_SpatialMaxPoolingCUDA_updateGradInput},
  {NULL, NULL}
};

static void cunn_SpatialMaxPoolingCUDA_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialMaxPoolingCUDA__, "nn");
  lua_pop(L,1);
}
