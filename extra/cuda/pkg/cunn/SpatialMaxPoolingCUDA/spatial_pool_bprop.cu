#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef SPATIAL_POOL_BPROP_CU
#define	SPATIAL_POOL_BPROP_CU

/*
 * Block size B_YxB_X
 * blockIdx.x determines pixel.x, image idx in batches of B_X*imgsPerThread
 * blockIdx.y determines pixel.y, filter idx in batches of B_Y*filtersPerThread
 * 
 * So each block does one output pixel for some number of images/filters.
 * 
 * threadIdx.x determines img idx
 * threadIdx.y determines filter idx
 * 
 * imgs:        (numFilters, imgPixels, numImages)
 * maxGrads:    (numFilters, numOutputs, numImages)
 * maxActs:    (numFilters, numOutputs, numImages)
 * target:      (numFilters, imgPixels, numImages)
 * 
 * numImages must be divisible by B_X*imgsPerThread
 * numFilters must be divisible by B_Y*filtersPerThread
 */

template<int B_Y, int B_X, int imgsPerThread, int filtersPerThread, bool add, bool checkCaseBounds>
__global__ void kLocalMaxUndo(float* imgs, float* maxGrads, float* maxActs, float* target, const int imgSize, const int numFilters,
                              const int numImages, const int subsX, const int startX, const int strideX, const int outputsX,
                              const float scaleTargets, const float scaleOutputs) {
    __shared__ float shImgs[B_Y*filtersPerThread][B_X*imgsPerThread];
    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockPxX = blockIdx.x / numImgBlocks;
    const int blockPxY = blockIdx.y / (numFilters/(B_Y*filtersPerThread));
    
    const int blockImgIdx = (blockIdx.x % numImgBlocks) * B_X * imgsPerThread;
    const int blockFilterIdx = (blockIdx.y % (numFilters/(B_Y*filtersPerThread))) * B_Y * filtersPerThread;
    
    const int blockPx = blockPxY * imgSize + blockPxX;
    const int numOutputs = outputsX * outputsX;
    const int imgPixels = imgSize * imgSize;

    const int startOutputY = blockPxY - startX < subsX ? 0 : 1 + (blockPxY - startX - subsX) / strideX;
    const int endOutputY = MIN(outputsX, 1 + (blockPxY - startX) / strideX);
    const int startOutputX = blockPxX - startX < subsX ? 0 : 1 + (blockPxX - startX - subsX) / strideX;
    const int endOutputX = MIN(outputsX, 1 + (blockPxX - startX) / strideX);
    
    const int imgIdx = blockImgIdx + threadIdx.x;
    
    imgs += ((blockFilterIdx + threadIdx.y) * imgPixels + blockPx) * numImages + imgIdx;
    maxGrads += ((blockFilterIdx + threadIdx.y) * numOutputs) * numImages 
            + imgIdx;
    maxActs += ((blockFilterIdx + threadIdx.y) * numOutputs) * numImages 
            + imgIdx;
    
    target += ((blockFilterIdx + threadIdx.y) * imgPixels + blockPx) * numImages + imgIdx;
    
    float prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for (int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[f][i] = 0;
        }
    }
    
    if  (blockPxX >= startX && blockPxX < startX + strideX * (outputsX-1) + subsX 
         && blockPxY >= startX && blockPxY < startX + strideX * (outputsX-1) + subsX) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || imgIdx + i * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    shImgs[threadIdx.y + B_Y * f][threadIdx.x + B_X * i] = imgs[f * B_Y * imgPixels * numImages + i * B_X];
                }
            }
        }
        for (int my = startOutputY; my < endOutputY; my++) {
            for (int mx = startOutputX; mx < endOutputX; mx++) {
                const int outputIdx = my * outputsX + mx;
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    if (!checkCaseBounds || imgIdx + i * B_X < numImages) {
                        #pragma unroll
                        for (int f = 0; f < filtersPerThread; f++) {
                            const float ma = maxActs[(f * B_Y * numOutputs + outputIdx) * numImages + i * B_X]; 
                            const float mg = maxGrads[(f * B_Y * numOutputs + outputIdx) * numImages + i * B_X];
                            const float img = shImgs[threadIdx.y + B_Y * f][threadIdx.x + B_X * i];

                            prod[f][i] += (img == ma) * mg;
                        }
                    }
                }
            }
        }
    }
    if (!add) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || imgIdx + i * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    target[f * B_Y * imgPixels * numImages + i * B_X] = prod[f][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || imgIdx + i * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    target[f * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * target[f * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[f][i];
                }
            }
        }
    }
}

/*
 * imgs:        (numFilters, imgPixels, numImages)
 * maxGrads:    (numFilters, numOutputs, numImages)
 * rMaxActs:    (numFilters, numOutputs, numImages)
 * target:      (numFilters, imgPixels, numImages)
 */

void spatialMaxPool_updateGradInput
(
 // raw pointers:
 float *images, float *maxgrads, float *maxacts, float *targets,
 // numImgColors == numFilters
 int numFilters, int imgSizeY, int imgSizeX, int numImages,
 // numModulesY == numModulesX == outputsX
 int numModulesY, int numModulesX, 
 // kH == kW == subsXs
 int filterSizeY, int filterSizeX, 
 // 0 == startX, dW == dH == strideX
 int paddingStart, int moduleStride, 
 // aux.
 float scaleTargets = 0, float scaleOutput = 1
) { 
  int imgPixels = imgSizeY * imgSizeX;
  int imgSize = int(sqrt(imgPixels)); 
  assert(imgSize * imgSize == imgPixels); /// TODO SQUARE !

  int subsX = filterSizeX;
  assert(filterSizeX == filterSizeY);
  
  int startX = paddingStart;
  int strideX = moduleStride;

  int outputsX = numModulesX;
  // int outputs = numModulesY * numModulesX;
  assert(numModulesY == numModulesX);  /// TODO SQUARE !

  assert(numFilters % 16 == 0);
  assert(strideX <= subsX);
    
  int imgsPerThread = numImages % 128 == 0 ? 4 : numImages % 64 == 0 ? 2 : 1;
  int checkCaseBounds = numImages % (32*imgsPerThread) != 0;
  dim3 threads(32, 4);
  dim3 blocks(DIVUP(numImages,32*imgsPerThread) * imgSize, (numFilters / (4 * 2)) * imgSize);
    
  if (imgsPerThread == 4) {
    if  (checkCaseBounds) {
      if (scaleTargets == 0 && scaleOutput == 1) {
	kLocalMaxUndo<4, 32, 4, 2, false, true><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								     imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      } else {
	kLocalMaxUndo<4, 32, 4, 2, true, true><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								    imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      }
    } else {
      if (scaleTargets == 0 && scaleOutput == 1) {
	kLocalMaxUndo<4, 32, 4, 2, false, false><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								      imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      } else {
	kLocalMaxUndo<4, 32, 4, 2, true, false><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								     imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      }
    }
  } else if (imgsPerThread == 2) {
    if  (checkCaseBounds) {
      if (scaleTargets == 0 && scaleOutput == 1) {
	kLocalMaxUndo<4, 32, 2, 2, false, true><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								     imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      } else {
	kLocalMaxUndo<4, 32, 2, 2, true, true><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								    imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      }
    } else {
      if (scaleTargets == 0 && scaleOutput == 1) {
	kLocalMaxUndo<4, 32, 2, 2, false, false><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								      imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      } else {
	kLocalMaxUndo<4, 32, 2, 2, true, false><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								     imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      }
    }
  } else {
    if  (checkCaseBounds) {
      if (scaleTargets == 0 && scaleOutput == 1) {
	kLocalMaxUndo<4, 32, 1, 2, false, true><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								     imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      } else {
	kLocalMaxUndo<4, 32, 1, 2, true, true><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								    imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      }
    } else {
      if (scaleTargets == 0 && scaleOutput == 1) {
	kLocalMaxUndo<4, 32, 1, 2, false, false><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								      imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      } else {
	kLocalMaxUndo<4, 32, 1, 2, true, false><<<blocks, threads>>>(images, maxgrads, maxacts, targets,
								     imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
      }
    }
  }

}

#endif	/* SPATIAL_POOL_BPROP_CU */
